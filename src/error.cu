#include <error.h>
#include <magma.h>


char * errors[] = {
		"Success!",
		"No device memory",
		"No parameter",
		"Invalid parameter",
		"Invalid config",
		"No file"
};

const char * cGetErrorString(natural value) {
	return errors[-value];
}

const char * mklGetErrorString(int value) {
	switch (value) {
		case -1010:
			return "LAPACK_WORK_MEMORY_ERROR";
		case -1011:
			return "LAPACK_TRANSPOSE_MEMORY_ERROR";
		default:
			return "Unknown MKL error";
	}
}

const char * curandGetErrorString(hiprandStatus_t value) {

	switch (value) {
		case HIPRAND_STATUS_SUCCESS:
			return "No error";
		case HIPRAND_STATUS_VERSION_MISMATCH:
			return "Header file and linked library version do not match.";
		case HIPRAND_STATUS_NOT_INITIALIZED:
			return "Generator not initialized.";
		case HIPRAND_STATUS_ALLOCATION_FAILED:
			return "Memory allocation failed.";
		case HIPRAND_STATUS_TYPE_ERROR:
			return "Generator is wrong type.";
		case HIPRAND_STATUS_OUT_OF_RANGE:
			return "Argument out of range.";
		case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
			return "Length requested is not a multple of dimension.";
		case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
			return "GPU does not have double precision required by MRG32k3a.";
		case HIPRAND_STATUS_LAUNCH_FAILURE:
			return "Kernel launch failure.";
		case HIPRAND_STATUS_PREEXISTING_FAILURE:
			return "Preexisting failure on library entry.";
		case HIPRAND_STATUS_INITIALIZATION_FAILED:
			return "Initialization of CUDA failed.";
		case HIPRAND_STATUS_ARCH_MISMATCH:
			return "rchitecture mismatch, GPU does not support requested feature.";
		case HIPRAND_STATUS_INTERNAL_ERROR:
			return "Internal library error.";
		default:
			return "Unknown error";

	}

}

const char * cublasv2GetErrorString(hipblasStatus_t value) {

	switch (value) {
		case HIPBLAS_STATUS_SUCCESS:
			return "No error";
		case HIPBLAS_STATUS_NOT_INITIALIZED:
			return "CUBLAS library not initialized";
		case HIPBLAS_STATUS_ALLOC_FAILED:
			return "Alloc failed";
		case HIPBLAS_STATUS_INVALID_VALUE:
			return "Invalid value";
		case HIPBLAS_STATUS_ARCH_MISMATCH:
			return "Architecture mismatch";
		case HIPBLAS_STATUS_MAPPING_ERROR:
			return "Mapping error";
		case HIPBLAS_STATUS_EXECUTION_FAILED:
			return "Execution failed";
		case HIPBLAS_STATUS_INTERNAL_ERROR:
			return "Internal error";
		default:
			return "Unknown error";
	}
}

const char * magmaGetErrorString(magma_err_t error) {
	return (magma_strerror(error));
}
